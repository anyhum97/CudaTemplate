#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "Reflection.cu"

void __global__ CudaSample(float* buf)
{
    /// <<<1, 128>>>

    const unsigned int block = blockIdx.x;
    const unsigned int thread = threadIdx.x;

    if(block > 0 || thread > 128)
    {
        return;
    }

    buf[thread] =  buf[thread]*2.0f;
}

int main()
{
    hipSetDevice(0);

    Reflection<float> buffer(128);

    for(int i=0; i<128; ++i)
    {
        buffer.host[i] = i+1;
    }

    buffer.Send();

    CudaSample<<<1, 128>>>(Device(buffer));

    buffer.Receive();

    return 0;
}