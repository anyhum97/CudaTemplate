#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "Reflection.cu"

////////////////////////////////////////////////////////////////////////

const unsigned int Width = 4;
const unsigned int Height = 4;

////////////////////////////////////////////////////////////////////////

Reflection<float> Buffer;   // [3][Width][Height];

////////////////////////////////////////////////////////////////////////

__inline__ __device__ unsigned int GetBufferIndex(const unsigned int dim, int x, int y)
{
    // Buffer[3][Width][Height];

    return dim*Width*Height + x*Height + y;
}

////////////////////////////////////////////////////////////////////////

void __global__ BufferAccess(float* Buffer)
{
    /// <<<Width, Height>>>

    const unsigned int block = blockIdx.x;
    const unsigned int thread = threadIdx.x;

    if(block >= Width || thread >= Height)
    {
        return;
    }

    Buffer[GetBufferIndex(0, block, thread)] = 1.0f;
    Buffer[GetBufferIndex(1, block, thread)] = 2.0f;
    Buffer[GetBufferIndex(2, block, thread)] = 3.0f;
}

////////////////////////////////////////////////////////////////////////

hipEvent_t start;
hipEvent_t stop;

////////////////////////////////////////////////////////////////////////

void CudaMalloc()
{
    hipSetDevice(0);

    Buffer = Malloc<float>(3*Width*Height);
}

void CudaFree()
{
    Free(Buffer);
}

////////////////////////////////////////////////////////////////////////

void Test()
{
    hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

    ////////////////////////////////////////////////////////////////////////

    BufferAccess<<<Width, Height>>>(Device(Buffer));

    ////////////////////////////////////////////////////////////////////////

    hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float time = 0;

	hipEventElapsedTime(&time, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

    ////////////////////////////////////////////////////////////////////////

    cout << time << "ms [OK]\n\n";

    ////////////////////////////////////////////////////////////////////////
}

void main()
{
    CudaMalloc();

    ////////////////////////////////////////////////////////////////////////

    Test();
    Receive(Buffer);
    Show(Buffer);

    ////////////////////////////////////////////////////////////////////////

    CudaFree();
}