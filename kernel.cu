#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "Reflection.cu"

int main()
{
    hipSetDevice(0);

    Reflection<float> test1(128);

    float* ptr = Host(test1);

    test1.Receive();

    ptr = Host(test1);

    return 0;
}