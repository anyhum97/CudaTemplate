#pragma once

#include "hip/hip_runtime.h"


#include <iostream>

template <typename Type>

struct Reflection
{
	Type* host = nullptr;
	Type* device = nullptr;

	unsigned int Size = 0;
	unsigned int IsValid = 0;
};

template <typename Type>

Reflection<Type> Malloc(const unsigned int count)
{
	Reflection<Type> reflection;

	const unsigned int size = count * sizeof(Type);

	if(hipMalloc(&reflection.device, size) != hipSuccess)
	{
		return reflection;
	}
	
	if(hipMemset(reflection.device, 0, size) != hipSuccess)
	{
		hipFree(reflection.device);
		reflection.device = nullptr;
		return reflection;
	}

	reflection.host = new Type[count];

	memset(reflection.host, 0, size);

	reflection.size = size;

	return reflection;
}



