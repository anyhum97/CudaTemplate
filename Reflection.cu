#pragma once

#include "hip/hip_runtime.h"
#include ""

#include <iostream>

////////////////////////////////////////////////////////////////////////

template <typename Type>

class Reflection
{
private:
	
	unsigned int Size = 0;

	void Clear()
	{
		host = nullptr;
		device = nullptr;
		
		Size = 0;
	}

public:

	Type* host = nullptr;
	Type* device = nullptr;

	Reflection::Reflection()
	{
		Clear();
	}

	Reflection::Reflection(const unsigned int count)
	{
		const unsigned int size = count * sizeof(Type);

		if(hipMalloc(&device, size) != hipSuccess)
		{
			Clear();
			return;
		}

		if(hipMemset(device, 0, size) != hipSuccess)
		{
			hipFree(device);
			Clear();
			return;
		}

		host = new Type[count];

		memset(host, 0, size);

		Size = size;
	}

	Reflection::Reflection(Type* buffer, const unsigned int count)
	{
		const unsigned int size = count * sizeof(Type);

		if(hipMalloc(&device, size) != hipSuccess)
		{
			Clear();
			return;
		}

		if(hipMemset(device, 0, size) != hipSuccess)
		{
			hipFree(device);
			Clear();
			return;
		}

		host = new Type[count];

		memcpy(host, buffer, size);

		Size = size;
	}

	Reflection::~Reflection()
	{
		Free();
	}

	bool IsValid()
	{
		if(Size == 0)
		{
			return false;
		}

		if(device == nullptr || host == nullptr)
		{
			return false;
		}

		return true;
	}

	void Free()
	{
		if(Size)
		{
			if(host != nullptr)
			{
				delete []host;
			}

			if(device != nullptr)
			{
				hipFree(device);
			}
		}

		Clear();
	}

	unsigned int GetSize()
	{
		return Size;
	}

	bool Send()
	{
		if(IsValid())
		{
			return hipMemcpy(device, host, Size, hipMemcpyHostToDevice) == hipSuccess;
		}

		return false;
	}

	bool Receive()
	{
		if(IsValid())
		{
			return hipMemcpy(host, device, Size, hipMemcpyDeviceToHost) == hipSuccess;
		}

		return false;
	}
};

////////////////////////////////////////////////////////////////////////

template <typename Type>

Type* Device(Reflection<Type>& reflection)
{
	return reflection.device;
}

////////////////////////////////////////////////////////////////////////

template <typename Type>

Type* Host(Reflection<Type>& reflection)
{
	return reflection.host;
}

////////////////////////////////////////////////////////////////////////






